#include "hip/hip_runtime.h"
//THIS PROGRAM FITS A SET OF DATA WITH THE RESONANCES YOU GIVE IT
//2/23/2017 
#include <fstream>

// GooFit stuff
#include "goofit/Application.h"
#include "goofit/Log.h"
#include "goofit/Variable.h" 
#include "goofit/fitting/FitManagerMinuit1.h"
#include "goofit/fitting/FitManagerMinuit2.h"
#include "goofit/PDFs/basic/PolynomialPdf.h" 
#include "goofit/PDFs/combine/AddPdf.h"
#include "goofit/UnbinnedDataSet.h"
#include "goofit/PDFs/physics/DP4Pdf.h"

#include <Minuit2/MnMigrad.h>
#include <Minuit2/MnPrint.h>
#include <Minuit2/FunctionMinimum.h>

using namespace std;
using namespace GooFit;

const fptype _mD0 = 1.8645; 
const fptype piPlusMass = 0.13957018;
const fptype piMinusMass = 0.13957018; 
const fptype kPlusMass = 0.493677; 
const fptype kMinusMass = 0.493677;
// Constants used in more than one PDF component. 

int main (int argc, char** argv) {

  // Place this at the beginning of main
  GooFit::Application app{"Optional discription", argc, argv};
  
  // Command line options can be added here.
  bool minuit1;
  app.add_flag("--minuit1", minuit1, "Use Minuit 1 instead of 2");

  bool minuit2; 
  app.add_flag("--minuit2", minuit2, "Use explicit minuit2 instance");

  std::string filename;
  app.add_option("filename", filename, "The file to run")
    ->required()
    ->check(CLI::ExistingFile);
  
  try {
      app.run();
  } catch(const GooFit::ParseError &e) {
      return app.exit(e);
  }


  Variable* m12 = new Variable("m12", 0, 3);
  Variable* m34 = new Variable("m34", 0, 3); 
  Variable* cos12 = new Variable("cos12", -1, 1);
  Variable* cos34 = new Variable("cos34", -1, 1);
  Variable* phi = new Variable("phi", 0.0, 2*M_PI);
  Variable* eventNumber = new Variable("eventNumber", 0, INT_MAX);

  double Amplitudes[30]; 

  std::vector<Variable*> vars;
  vars.push_back(m12);
  vars.push_back(m34);
  vars.push_back(cos12);
  vars.push_back(cos34);
  vars.push_back(phi);
  vars.push_back(eventNumber); 
  UnbinnedDataSet currData(vars); 

  unsigned int MCevents = 0;
//Load in nTuple and give it to currData/addevent 
  fstream input(filename, std::ios_base::in);
  while(input >> *m12 >> *m34 >> *cos12 >> *cos34 >> *phi){
    //if(!*m12 || !*m34 || !*cos12 || !*cos34 || !*phi)
    //    continue;
    *eventNumber = MCevents++; 
    currData.addEvent();
  }

  printf("done reading in %i events\n", MCevents );

  DecayInfo_DP* DKKPP_DI = new DecayInfo_DP();
  DKKPP_DI->meson_radius =1.5;
  DKKPP_DI->particle_masses.push_back(_mD0);
  DKKPP_DI->particle_masses.push_back(piPlusMass);
  DKKPP_DI->particle_masses.push_back(piMinusMass);
  DKKPP_DI->particle_masses.push_back(kPlusMass);
  DKKPP_DI->particle_masses.push_back(kMinusMass);

  //Need to add K1(1270), phi, kstar0 and kstar0bar?? (this might be K1430 christoph has listed), k
  Variable* RhoMass  = new Variable("rho_mass" , 0.77526);
  Variable* RhoWidth = new Variable("rho_width", 0.1478 ); 
  Variable* K11270M = new Variable("K11270M", 1.272); 
  Variable* K11270W = new Variable("K11270W", 0.09); 
  Variable* phi1020M = new Variable("phi1020M", 1.019); 
  Variable* phi1020W = new Variable("phi1020W", 0.004); 
  Variable* K1430M   = new Variable("K1430M"   , 1.425  );//not used?
  Variable* K1430W   = new Variable("K1430W"   , 0.27   );
  Variable* Kstar1410M = new Variable("Kstar1410M" ,   1.414);
  Variable* Kstar1410W = new Variable("Kstar1410W",   0.232);
  //Not Used
  Variable* FZeroMass    = new Variable("f600M"    , 0.519  );
  Variable* FZeroWidth    = new Variable("f600W"    , 0.454  ); 
  Variable* a1M      = new Variable("a1M"      , 1.23   );
  Variable* a1W      = new Variable("a1W"      , 0.42   ); 
  Variable* K1M      = new Variable("K1M"      , 1.272  );
  Variable* K1W      = new Variable("K1W"      , 0.09   ); 
  Variable* Kstar892M   = new Variable("Kstar892M"   , 0.89581);
  Variable* Kstar892W   = new Variable("Kstar892W"   , 0.0474 ); 
  

  //Spin factors: we have two due to the bose symmetrization of the two pi+
  //K11270->Kstar0 (2?)----
  //K11270->rho (2?)----- 
  //Kstar1410-> Kstar0 (2?)-----
  //phi rho (D and S)
  //phi  (s) non resonant?
  // non resonant?

/////////////----------------------NOTE----------------
// p1-> pi+
// p2-> pi-
// p3-> K+
// p4-> K-

  std::vector<SpinFactor*> SFK1P2Kstar1430;//K1(1270)+(Kstar0 pi+)K- 
  SFK1P2Kstar1430.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, 3,0,1,2));
  SFK1P2Kstar1430.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1,2,0,1,3));

  std::vector<SpinFactor*> SFK1M2Kstar1430;//K1(1270)-(Kstar0bar pi-)K+ 
  SFK1M2Kstar1430.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, 2,1,0,3)); 
  SFK1M2Kstar1430.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1,3,1,0,2));

  std::vector<SpinFactor*> SFK1P2Kstar;//K1(1270)+(Kstar0 pi+)K- 
  SFK1P2Kstar.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, 3,0,1,2));
  SFK1P2Kstar.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1,2,0,1,3));

  std::vector<SpinFactor*> SFK1M2Kstar;//K1(1270)-(Kstar0bar pi-)K+ 
  SFK1M2Kstar.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, 2,1,0,3));
  SFK1M2Kstar.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1,3,1,0,2));

  std::vector<SpinFactor*> SFK1P2Rho;//K1(1270)+(rho K+)K- 
  SFK1P2Rho.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, 3,2,0,1)); 
  SFK1P2Rho.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1,1,2,0,3));

  std::vector<SpinFactor*> SFK1M2Rho;//K1(1270)-(rho K-)K+ 
  SFK1M2Rho.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, 2,3,1,0)); 
  SFK1M2Rho.push_back( new SpinFactor("SF",SF_4Body::FF_123_4_L1,0,3,1,2));


  std::vector<SpinFactor*> SFKstarP2Kstar;//Kstar(1410)+(Kstar0 pi+)K- 
  SFKstarP2Kstar.push_back( new SpinFactor("SF", SF_4Body::DtoPP1_PtoVP2_VtoP3P4, 3,0,1,2));
  SFKstarP2Kstar.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1,2,0,1,3));

  std::vector<SpinFactor*> SFKstarM2Kstar;//Kstar(1410)-(Kstar0 pi-)K+ 
  SFKstarM2Kstar.push_back( new SpinFactor("SF", SF_4Body::DtoPP1_PtoVP2_VtoP3P4,2,1,0,3));  
  SFKstarM2Kstar.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1,3,1,0,2));

  std::vector<SpinFactor*> SFKstarKstarS;
  SFKstarKstarS.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_S, 0,3,1,2));  

  std::vector<SpinFactor*> SFKstarKstarP;
  SFKstarKstarP.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_P, 0,3,1,2));  
  SFKstarKstarP.push_back(new SpinFactor("SF",SF_4Body::FF_12_34_L1, 0,3,1,2));
  
  std::vector<SpinFactor*> SFKstarKstarD;
  SFKstarKstarD.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_D, 0,3,1,2));  
  SFKstarKstarD.push_back(new SpinFactor("SF",SF_4Body::FF_12_34_L2, 0,3,1,2));
 

  std::vector<SpinFactor*> SFPhiRhoS; 
  SFPhiRhoS.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_S, 2,3,0,1)); 
  //SFPhiRhoD.push_back(new SpinFactor("SF",SF_4Body::FF_12_34_L2, 2,3,0,1));
 

  std::vector<SpinFactor*> SFPhiRhoP; 
  SFPhiRhoP.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_P, 2,3,0,1)); 
  SFPhiRhoP.push_back(new SpinFactor("SF",SF_4Body::FF_12_34_L1, 2,3,0,1));
 

  std::vector<SpinFactor*> SFPhiRhoD;
  SFPhiRhoD.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_D, 2,3,0,1)); 
  SFPhiRhoD.push_back(new SpinFactor("SF",SF_4Body::FF_12_34_L2, 2,3,0,1));
 
  std::vector<SpinFactor*> SFPhiFZero;
  SFPhiFZero.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_D, 2,3,0,1)); 
  SFPhiFZero.push_back( new SpinFactor("SF",SF_4Body::FF_12_34_L2,2,3,0,1));

 // std::vector<SpinFactor*> SFPhipipi; 
 // SFPhipipi.push_back( new SpinFactor("SF", SF_4Body::DtoVP1P2_VtoP3P4, 0,1,2,3)); 

  std::vector<SpinFactor*> SFNonRes1;
  SFNonRes1.push_back( new SpinFactor("SF", SF_4Body::ONE, 0,1,2,3)); 

  std::vector<SpinFactor*> SFNonRes2; 
  SFNonRes2.push_back( new SpinFactor("SF", SF_4Body::ONE, 1,2,0,3));   

  //////////////////////////////////////////////////////////////////////////////////////////////////

  std::vector<Lineshape*> LSK1P2Kstar1430;
  LSK1P2Kstar1430.push_back( new Lineshape("K1(1270)+", K11270M, K11270W, 1, M_23_1, LS::BW, FF::BL2) );    
  LSK1P2Kstar1430.push_back( new Lineshape("Kstar(1430)", K1430M, K1430W, 0, M_23, LS::BW, FF::BL2) ); 

  std::vector<Lineshape*> LSK1M2Kstar1430; 
  LSK1M2Kstar1430.push_back( new Lineshape("K1(1270)-", K11270M, K11270W, 1, M_14_2, LS::BW, FF::BL2) ); 
  LSK1M2Kstar1430.push_back( new Lineshape("Kstar(1430)", K1430M, K1430W,0, M_14, LS::BW, FF::BL2) );  

  std::vector<Lineshape*> LSK1P2Kstar;
  LSK1P2Kstar.push_back( new Lineshape("K1(1270)+", K11270M, K11270W, 0, M_23_1, LS::BW, FF::BL2) );    
  LSK1P2Kstar.push_back( new Lineshape("Kstar(892)", Kstar892M, Kstar892W, 1, M_23, LS::BW, FF::BL2) ); 

  std::vector<Lineshape*> LSK1M2Kstar; 
  LSK1M2Kstar.push_back( new Lineshape("K1(1270)-", K11270M, K11270W, 0, M_14_2, LS::BW, FF::BL2) ); 
  LSK1M2Kstar.push_back( new Lineshape("Kstar(892)", Kstar892M, Kstar892W,1, M_14, LS::BW, FF::BL2) );  

  std::vector<Lineshape*> LSK1P2Rho;
  LSK1P2Rho.push_back( new Lineshape("K1(1270)+", K11270M, K11270W, 0, M_12_3, LS::BW, FF::BL2) );
  LSK1P2Rho.push_back( new Lineshape("Rho(770)", RhoMass, RhoWidth, 1, M_12, LS::BW, FF::BL2) );   

  std::vector<Lineshape*> LSK1M2Rho; 
  LSK1M2Rho.push_back( new Lineshape("K1(1270)-", K11270M, K11270W, 0, M_12_4, LS::BW, FF::BL2) ); 
  LSK1M2Rho.push_back( new Lineshape("Rho(770)", RhoMass, RhoWidth, 1, M_12, LS::BW, FF::BL2) ); 
 
  std::vector<Lineshape*> LSKstarP2Kstar;
  LSKstarP2Kstar.push_back( new Lineshape("Kstar1410", Kstar1410M, Kstar1410W, 1, M_23_1, LS::BW, FF::BL2) ); 
  LSKstarP2Kstar.push_back( new Lineshape("Kstar892", Kstar892M, Kstar892W, 1, M_23, LS::BW, FF::BL2) );  

  std::vector<Lineshape*> LSKstarM2Kstar; 
  LSKstarM2Kstar.push_back( new Lineshape("Kstar1410", Kstar1410M, Kstar1410W, 1, M_14_2, LS::BW, FF::BL2) ); 
  LSKstarM2Kstar.push_back( new Lineshape("Kstar(892)", Kstar892M, Kstar892W, 1, M_14, LS::BW, FF::BL2) ); 
   
  std::vector<Lineshape*> LSKstarKstarbarS; 
  LSKstarKstarbarS.push_back( new Lineshape("Kstar(892) ", Kstar892M, Kstar892W, 1, M_23, LS::BW, FF::BL2) ); 
  LSKstarKstarbarS.push_back( new Lineshape("Kstarbar(892)", Kstar892M, Kstar892W, 1, M_14, LS::BW, FF::BL2) ); 
  
  std::vector<Lineshape*> LSKstarKstarbarP; 
  LSKstarKstarbarP.push_back( new Lineshape("Kstar(892) ", Kstar892M, Kstar892W, 1, M_23, LS::BW, FF::BL2) ); 
  LSKstarKstarbarP.push_back( new Lineshape("Kstarbar(892)", Kstar892M, Kstar892W, 1, M_14, LS::BW, FF::BL2) ); 
  
  std::vector<Lineshape*> LSKstarKstarbarD; 
  LSKstarKstarbarD.push_back( new Lineshape("Kstar(892)", Kstar892M, Kstar892W, 1, M_23, LS::BW, FF::BL2) ); 
  LSKstarKstarbarD.push_back( new Lineshape("Kstarbar(892)", Kstar892M, Kstar892W, 1, M_14, LS::BW, FF::BL2) ); 
  
  std::vector<Lineshape*> LSPhiRhoS; 
  LSPhiRhoS.push_back( new Lineshape("Phi(1020) ", phi1020M, phi1020W, 1, M_34, LS::BW, FF::BL2) ); 
  LSPhiRhoS.push_back( new Lineshape("Rho(770)", RhoMass, RhoWidth, 1, M_12, LS::BW, FF::BL2) ); 

  std::vector<Lineshape*> LSPhiRhoP; 
  LSPhiRhoP.push_back( new Lineshape("Phi(1020) ", phi1020M, phi1020W, 1, M_34, LS::BW, FF::BL2) ); 
  LSPhiRhoP.push_back( new Lineshape("Rho(770) ", RhoMass, RhoWidth, 1, M_12, LS::BW, FF::BL2) ); 


  std::vector<Lineshape*> LSPhiRhoD; 
  LSPhiRhoD.push_back( new Lineshape("Phi ", phi1020M, phi1020W, 1, M_34, LS::BW, FF::BL2) ); 
  LSPhiRhoD.push_back( new Lineshape("Rho ", RhoMass, RhoWidth, 1, M_12, LS::BW, FF::BL2) ); 

  std::vector<Lineshape*> LSPhiFZero; 
  LSPhiFZero.push_back( new Lineshape("Phi ", phi1020M, phi1020W, 1, M_34, LS::BW, FF::BL2) ); 
  LSPhiFZero.push_back( new Lineshape("Rho ", FZeroMass, FZeroWidth, 0, M_12, LS::BW, FF::BL2) ); 



//  std::vector<Lineshape*> LSPhiRhoP;  // std::vector<Lineshape*> LSPhipipi; 
 // LSPhipipi.push_back( new Lineshape("Phi ",phi1020M,phi1020W, 1, M_34, LS::BW, FF::BL2) ); 
 // LSPhipipi.push_back( new Lineshape("pipi ", new Variable("NR5", 0.0), new Variable("NR6",0.0), 1, M_12, LS::BW, FF::BL2) ); 

  std::vector<Lineshape*> LSNonRes1; 
  LSNonRes1.push_back( new Lineshape("NonRes1 ", new Variable("NR1", 0.0), new Variable("NR2", 0.0),1, M_34_2, LS::nonRes, FF::BL2) ); 

  std::vector<Lineshape*> LSNonRes2; 
  LSNonRes2.push_back( new Lineshape("NonRes2 ", new Variable("NR3", 0.0), new Variable("NR4", 0.0),1, M_34_2, LS::nonRes, FF::BL2) ); 
 /////////////////////////////////////////////////////////////////////////////////////////////////

 // Amplitude* AMP_K1P2Kstar = new Amplitude("K1(1270)+(Kstar0 pi+)K- ", new Variable("amp_real1", -0.1, 0.001,0,0), new Variable("amp_imag1", 0.1, 0.001, 0,0), LSK1P2Kstar, SFK1P2Kstar, 1);

  Amplitude* AMP_K1P2Kstar1430 = new Amplitude("K1(1270)+(Kstar1430) pi+)K- ",new Variable("AmPK1P2Kstar1430_R", -0.1, 0.001, 0,0),new Variable("AmpK1P2Kstar1430_I", 0.1, 0.001, 0, 0), LSK1P2Kstar1430, SFK1P2Kstar1430, 1);  

  Amplitude* AMP_K1M2Kstar1430 = new Amplitude("K1(1270)-(Kstar1430 pi-)K+ ", new Variable("AmpK1M2Kstar1430_R", -0.1, 0.001,0,0), new Variable("AmpK1M2Kstar1430_I", 0.1, 0.001, 0,0), LSK1M2Kstar1430, SFK1M2Kstar1430, 1); 

  Amplitude* AMP_K1P2Kstar = new Amplitude("K1(1270)+(Kstar0) pi+)K- ",new Variable("AmPK1P2Kstar_R", -0.1, 0.001, 0,0),new Variable("AmpK1P2Kstar_I", 0.1, 0.001, 0, 0), LSK1P2Kstar, SFK1P2Kstar, 1);  

  Amplitude* AMP_K1M2Kstar = new Amplitude("K1(1270)-(Kstar0 pi-)K+ ", new Variable("AmpK1M2Kstar_R", -0.1, 0.001,0,0), new Variable("AmpK1M2Kstar_I", 0.1, 0.001, 0,0), LSK1M2Kstar, SFK1M2Kstar, 1); 

  Amplitude* AMP_K1P2Rho = new Amplitude("K1(1270)+(Rho K+) K- ", new Variable("AmpK1P2Rho_R", -0.1, 0.001,0,0), new Variable("AmpK1P2Rho_I", 0.1, 0.001, 0,0), LSK1P2Rho, SFK1P2Rho, 1);

  Amplitude* AMP_K1M2Rho = new Amplitude("K1(1270)-(Rho k-)K+ ", new Variable("AmpK1M2Rho_R", -0.1, 0.001,0,0), new Variable("AmpK1M2Rho_I", 0.1, 0.001, 0,0), LSK1M2Rho, SFK1M2Rho, 1);

  Amplitude* AMP_KstarP2Kstar = new Amplitude("Kstar(1410)+(Kstar pi+)K- ", new Variable("AmpKstarP2Kstar_R", -0.1, 0.001,0,0), new Variable("AmpKstarP2Kstar_I", 0.1, 0.001, 0,0), LSKstarP2Kstar, SFKstarP2Kstar, 1);

  Amplitude* AMP_KstarM2Kstar = new Amplitude("Kstar(1410)-(Kstarbar pi-) K+ ", new Variable("AMpKstarM2Kstar_R", -0.1, 0.001,0,0), new Variable("AmpKstarM2Kstar_I", 0.1, 0.001, 0,0), LSKstarM2Kstar, SFKstarM2Kstar, 1);

  Amplitude* AMP_KstarKstarbarS = new Amplitude("KstarKstarS ", new Variable("AmpKstarKstarbarS_R", -0.1, 0.001,0,0), new Variable("AmpKstarKstarbarS_I",  -0.1,0.001, 0,0), LSKstarKstarbarS, SFKstarKstarS, 1);


  Amplitude* AMP_KstarKstarbarP = new Amplitude("KstarKstarP ", new Variable("AmpKstarKstarbarP_R", -0.1, 0.001,0,0), new Variable("AmpKstarKstarbarP_I",  -0.1,0.001, 0,0), LSKstarKstarbarP, SFKstarKstarP, 1);


  Amplitude* AMP_KstarKstarbarD = new Amplitude("KstarKstarD ", new Variable("AmpKstarKstarbarD_R", -0.1, 0.001,0,0), new Variable("AmpKstarKstarbarD_I",  -0.1,0.001, 0,0), LSKstarKstarbarD, SFKstarKstarD, 1);

  Amplitude* AMP_PhiRhoSFix = new Amplitude("PhiRhoS", new Variable("AmpPhiRhoS_R", 1), new Variable("AmpPhiRhoS_I", 0), LSPhiRhoS, SFPhiRhoS, 1);

  Amplitude* AMP_PhiRhoP = new Amplitude("PhiRhoP", new Variable("AmpPhiRhoP_R", -0.1, 0.001,0,0), new Variable("AmpPhiRhoP_I", 0.1, 0.001, 0,0), LSPhiRhoP, SFPhiRhoP, 1);


  Amplitude* AMP_PhiRhoD = new Amplitude("PhiRhoD", new Variable("AmpPhiRhoD_R", -0.1, 0.001,0,0), new Variable("AmpPhiRhoD_I", 0.1, 0.001, 0,0), LSPhiRhoD, SFPhiRhoD, 1);

 
  Amplitude* AMP_PhiFZero = new Amplitude("PhiFZero", new Variable("AmpPhiFZero_R", -0.1, 0.001,0,0), new Variable("AmpPhiFZero_I", 0.1, 0.001, 0,0), LSPhiFZero, SFPhiFZero, 1);

// Amplitude* AMP_PhiPiPi = new Amplitude("PhiPiPi", new Variable("amp_real10", -0.1, 0.001,0,0), new Variable("amp_imag10", 0.1, 0.001, 0,0), LSPhipipi, SFPhipipi, 1);

  Amplitude* AMP_NonRes1 = new Amplitude("NonRes1", new Variable("amp_real11", -0.1, 0.001,0,0), new Variable("amp_imag11", 0.1, 0.001, 0,0), LSNonRes1, SFNonRes1, 1);

  Amplitude* AMP_NonRes2 = new Amplitude("NonRes2", new Variable("amp_real12", -0.1, 0.001,0,0), new Variable("amp_imag12", 0.1, 0.001, 0,0), LSNonRes1, SFNonRes2, 1);


  DKKPP_DI->amplitudes.push_back(AMP_K1P2Kstar1430);
  DKKPP_DI->amplitudes.push_back(AMP_K1M2Kstar1430);
  DKKPP_DI->amplitudes.push_back(AMP_K1P2Kstar);
  DKKPP_DI->amplitudes.push_back(AMP_K1M2Kstar);
  DKKPP_DI->amplitudes.push_back(AMP_K1P2Rho);
  DKKPP_DI->amplitudes.push_back(AMP_K1M2Rho);
  DKKPP_DI->amplitudes.push_back(AMP_KstarP2Kstar);
  DKKPP_DI->amplitudes.push_back(AMP_KstarM2Kstar);
  DKKPP_DI->amplitudes.push_back(AMP_KstarKstarbarS); 
  DKKPP_DI->amplitudes.push_back(AMP_KstarKstarbarP);
  DKKPP_DI->amplitudes.push_back(AMP_KstarKstarbarD);
  DKKPP_DI->amplitudes.push_back(AMP_PhiRhoSFix);
  DKKPP_DI->amplitudes.push_back(AMP_PhiRhoP);
  DKKPP_DI->amplitudes.push_back(AMP_PhiRhoD);
  DKKPP_DI->amplitudes.push_back(AMP_PhiFZero); 
  //DKKPP_DI->amplitudes.push_back(AMP_PhiPiPi);
  //DKKPP_DI->amplitudes.push_back(AMP_NonRes1);
  //DKKPP_DI->amplitudes.push_back(AMP_NonRes2);




  for (auto res = LSK1P2Kstar1430.begin(); res != LSK1P2Kstar1430.end(); ++res) {
    (*res)->setParameterConstantness(true); 
  }

  for (auto res = LSK1M2Kstar1430.begin(); res != LSK1M2Kstar1430.end(); ++res) {
    (*res)->setParameterConstantness(true); 
  }

  for (auto res = LSK1P2Kstar.begin(); res != LSK1P2Kstar.end(); ++res) {
    (*res)->setParameterConstantness(true); 
  }

  for (auto res = LSK1M2Kstar.begin(); res != LSK1M2Kstar.end(); ++res) {
    (*res)->setParameterConstantness(true); 
  }

  for (auto res = LSK1P2Rho.begin(); res != LSK1P2Rho.end(); ++res) {
    (*res)->setParameterConstantness(true); 
  }

  for (auto res = LSK1M2Rho.begin(); res != LSK1M2Rho.end(); ++res) {
    (*res)->setParameterConstantness(true); 
  }
  for (auto res = LSKstarP2Kstar.begin(); res != LSKstarP2Kstar.end(); ++res) {
    (*res)->setParameterConstantness(true); 
  }

  for (auto res = LSKstarM2Kstar.begin(); res != LSKstarM2Kstar.end(); ++res) {
    (*res)->setParameterConstantness(true); 
  }

  for (auto res = LSKstarKstarbarS.begin(); res != LSKstarKstarbarS.end(); ++res) {
    (*res)->setParameterConstantness(true); 
  }


  for (auto res = LSKstarKstarbarP.begin(); res != LSKstarKstarbarP.end(); ++res) {
    (*res)->setParameterConstantness(true); 
  }


  for (auto res = LSKstarKstarbarD.begin(); res != LSKstarKstarbarD.end(); ++res) {
    (*res)->setParameterConstantness(true); 
  }

  for (auto res = LSPhiRhoS.begin(); res != LSPhiRhoS.end(); ++res) {
    (*res)->setParameterConstantness(true); 
  }


  for (auto res = LSPhiRhoP.begin(); res != LSPhiRhoP.end(); ++res) {
    (*res)->setParameterConstantness(true); 
  }


  for (auto res = LSPhiRhoD.begin(); res != LSPhiRhoD.end(); ++res) {
    (*res)->setParameterConstantness(true); 
  }

  for (auto res = LSPhiFZero.begin(); res != LSPhiFZero.end(); ++res) {
    (*res)->setParameterConstantness(true); 
  }

 // for (auto res = LSPhipipi.begin(); res != LSPhipipi.end(); ++res) {
 //   (*res)->setParameterConstantness(true); 
 // }


 // for (auto res = LSNonRes1.begin(); res != LSNonRes1.end(); ++res) {
//    (*res)->setParameterConstantness(true); 
//  }



 // for (auto res = LSNonRes2.begin(); res != LSNonRes2.end(); ++res) {
  //  (*res)->setParameterConstantness(true); 
 // }



  Variable* constantOne = new Variable("constantOne", 1); 
  Variable* constantZero = new Variable("constantZero", 0);

  vector<Variable*> observables;
  vector<Variable*> coefficients; 
  vector<Variable*> offsets;

  observables.push_back(m12);
  observables.push_back(m34);
  observables.push_back(cos12);
  observables.push_back(cos34);
  observables.push_back(phi);
  observables.push_back(eventNumber);
  offsets.push_back(constantZero);
  offsets.push_back(constantZero);
  coefficients.push_back(constantOne); 

  PolynomialPdf* eff = new PolynomialPdf("constantEff", observables, coefficients, offsets, 0);
  DPPdf* dp = new DPPdf("test", observables, DKKPP_DI, eff,1e6);

  Variable* constant = new Variable("constant", 0.1); 
  Variable* constant2 = new Variable("constant2", 1.0); 
  vars.clear();
  vars.push_back(constant);
  PolynomialPdf backgr("backgr", m12, vars); 
  AddPdf* signal = new AddPdf("signal",constant2,dp, &backgr);

  signal->setData(&currData);
  dp->setDataSize(currData.getNumEvents(), 6); 

  if(minuit1) {
    GooFit::FitManagerMinuit1 datapdf(signal);
    datapdf.useHesseBefore(false);
    datapdf.fit();
    return 0; 
  } else {

    if(minuit2) {
      GooFit::Params upar{*signal};
      GooFit::FCN fcn{upar};
      Minuit2::MnPrint::SetLevel(3);
      Minuit2::MnMigrad migrad{fcn, upar};
      Minuit2::FunctionMinimum min = migrad(10000);
      cout << min << endl;
      return 0;
    } else {
      GooFit::FitManagerMinuit2 datapdf(signal);
      datapdf.setMaxCalls(10000);
      datapdf.fit();
      return datapdf; 
    }
  }
}
