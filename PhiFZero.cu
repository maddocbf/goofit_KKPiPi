#include "hip/hip_runtime.h"
//THIS PROGRAM GENERATES MONTECARLO DATA GIVEN AN AMPLITUDE MODEL


//ROOT
#include <TFile.h>
#include <TTree.h>
#include <iostream>
#include <string>

// GooFit stuff
#include "goofit/Variable.h" 
#include "goofit/PDFs/PolynomialPdf.h" 
#include "goofit/UnbinnedDataSet.h"
#include "goofit/PDFs/DP4Pdf.h"
#include "goofit/PDFs/TruthResolution_Aux.h" 
#include "goofit/PDFs/Tddp4Pdf.h"
#include <thrust/count.h>
#include <fstream>
#include <iomanip>
#include <numeric>
#include <algorithm>
#include <random>
#include <ctime>
#include <functional>
#include <mcbooster/functors/FlagAcceptReject.h>

using namespace std;

// Constants used in more than one PDF component. 
const fptype _mD0 = 1.8645; 
const fptype piPlusMass = 0.13957018;
const fptype piMinusMass = 0.13957018; 
const fptype kPlusMass = 0.493677; 
const fptype kMinusMass = .493677;
int main (int argc, char** argv) {
  DecayInfo_DP* DKKPP_DI = new DecayInfo_DP();
  DKKPP_DI->meson_radius =1.5;
  DKKPP_DI->particle_masses.push_back(_mD0);
  DKKPP_DI->particle_masses.push_back(piPlusMass);
  DKKPP_DI->particle_masses.push_back(piMinusMass);
  DKKPP_DI->particle_masses.push_back(kPlusMass);
  DKKPP_DI->particle_masses.push_back(kMinusMass);
 
  Variable* RhoMass  =  new Variable("rho_mass", 0.77526);
  Variable* RhoWidth =  new Variable("rho_width", 0.1478); 
  Variable* Kstar892M   =   new Variable("K892M", 0.89581);
  Variable* Kstar892W   =   new Variable("K892W", 0.0474); 
  Variable* FZeroMass  =    new Variable("f600M", 0.519);
  Variable* FZeroWidth  =    new Variable("f600W", 0.454); 
  Variable* a1M  =      new Variable("a1M", 1.237);
  Variable* a1W  =      new Variable("a1W", 0.526); 
  

Variable* K11270M  = new Variable("K1_1270M", 1.272);
  Variable* K11270W  = new Variable("K1_1270W", 0.09); 
  Variable* K1430M  = new Variable("K0_1430M", 1.425);
  Variable* K1430W  = new Variable("K0_1430W", 0.27);

  Variable* Kstar1410M    = new Variable("K1410M", 1.414);
  Variable* Kstar1410W    = new Variable("K1410W", 0.232); 

//  Variable* rho1450M  = new Variable("rho1450M", 1.465);
//  Variable* rho1450W  = new Variable("rho1450W", 0.400); 

 // Variable* K1460M    = new Variable("K1460M", 1.351);
 // Variable* K1460W    = new Variable("K1460W", 0.281); 

 // Variable* f0_1370M  = new Variable("f0_1370M", 1.350);
 // Variable* f0_1370W  = new Variable("f0_1370W", 0.35); 

  Variable* K1_1400M  = new Variable("K1_1400M", 1.403);
  Variable* K1_1400W  = new Variable("K1_1400W", 0.174); 

  Variable* K2_1430M  = new Variable("K2_1430M", 1.4256);
  Variable* K2_1430W  = new Variable("K2_1430W", 0.0985); 
  
  Variable* phi1020M = new Variable("phi1020M", 1.019); 
  Variable* phi1020W = new Variable("phi1020W", 0.004); 

  std::vector<Variable*> LassVars;
  LassVars.push_back( new Variable("lass_a",2.07) );
  LassVars.push_back( new Variable("lass_r",3.32) );
  LassVars.push_back( new Variable("lass_pf",0.0) );
  LassVars.push_back( new Variable("lass_pr",0.0) );
  LassVars.push_back( new Variable("lass_F",1.0) );
 ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

  std::vector<SpinFactor*> SFK1P2Kstar1430;//K1(1270)+(Kstar0 pi+)K- 
  SFK1P2Kstar1430.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, 3,0,1,2));
  SFK1P2Kstar1430.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1,2,0,1,3));

  std::vector<SpinFactor*> SFK1M2Kstar1430;//K1(1270)-(Kstar0bar pi-)K+ 
  SFK1M2Kstar1430.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, 2,1,0,3)); 
  SFK1M2Kstar1430.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1,3,1,0,2));

  std::vector<SpinFactor*> SFK1P2Kstar;//K1(1270)+(Kstar0 pi+)K- 
  SFK1P2Kstar.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, 3,0,1,2));
  SFK1P2Kstar.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1,2,0,1,3));

  std::vector<SpinFactor*> SFK1M2Kstar;//K1(1270)-(Kstar0bar pi-)K+ 
  SFK1M2Kstar.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, 2,1,0,3));
  SFK1M2Kstar.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1,3,1,0,2));

  std::vector<SpinFactor*> SFK1P2Rho;//K1(1270)+(rho K+)K- 
  SFK1P2Rho.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, 3,2,0,1)); 
  SFK1P2Rho.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1,1,2,0,3));

  std::vector<SpinFactor*> SFK1M2Rho;//K1(1270)-(rho K-)K+ 
  SFK1M2Rho.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, 2,3,1,0)); 
  SFK1M2Rho.push_back( new SpinFactor("SF",SF_4Body::FF_123_4_L1,0,3,1,2));


  std::vector<SpinFactor*> SFKstarP2Kstar;//Kstar(1410)+(Kstar0 pi+)K- 
  SFKstarP2Kstar.push_back( new SpinFactor("SF", SF_4Body::DtoPP1_PtoVP2_VtoP3P4, 3,0,1,2));
  SFKstarP2Kstar.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1,2,0,1,3));

  std::vector<SpinFactor*> SFKstarM2Kstar;//Kstar(1410)-(Kstar0 pi-)K+ 
  SFKstarM2Kstar.push_back( new SpinFactor("SF", SF_4Body::DtoPP1_PtoVP2_VtoP3P4,2,1,0,3));  
  SFKstarM2Kstar.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1,3,1,0,2));

  std::vector<SpinFactor*> SFKstarKstarS;
  SFKstarKstarS.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_S, 0,3,1,2));  

  std::vector<SpinFactor*> SFKstarKstarP;
  SFKstarKstarP.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_P, 0,3,1,2));  
  SFKstarKstarP.push_back(new SpinFactor("SF",SF_4Body::FF_12_34_L1, 0,3,1,2));
  
  std::vector<SpinFactor*> SFKstarKstarD;
  SFKstarKstarD.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_D, 0,3,1,2));  
  SFKstarKstarD.push_back(new SpinFactor("SF",SF_4Body::FF_12_34_L2, 0,3,1,2));
 

  std::vector<SpinFactor*> SFPhiRhoS; 
  SFPhiRhoS.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_S, 2,3,0,1)); 
  //SFPhiRhoD.push_back(new SpinFactor("SF",SF_4Body::FF_12_34_L2, 2,3,0,1));
 

  std::vector<SpinFactor*> SFPhiRhoP; 
  SFPhiRhoP.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_P, 2,3,0,1)); 
  SFPhiRhoP.push_back(new SpinFactor("SF",SF_4Body::FF_12_34_L1, 2,3,0,1));
 

  std::vector<SpinFactor*> SFPhiRhoD;
  SFPhiRhoD.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_D, 2,3,0,1)); 
  SFPhiRhoD.push_back(new SpinFactor("SF",SF_4Body::FF_12_34_L2, 2,3,0,1));
 
  std::vector<SpinFactor*> SFPhiFZero;
  SFPhiFZero.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_D, 2,3,0,1)); 
  SFPhiFZero.push_back( new SpinFactor("SF",SF_4Body::FF_12_34_L2,2,3,1,2));

 // std::vector<SpinFactor*> SFPhipipi; 
 // SFPhipipi.push_back( new SpinFactor("SF", SF_4Body::DtoVP1P2_VtoP3P4, 0,1,2,3)); 

  std::vector<SpinFactor*> SFNonRes1;
  SFNonRes1.push_back( new SpinFactor("SF", SF_4Body::ONE, 0,1,2,3)); 

  std::vector<SpinFactor*> SFNonRes2; 
  SFNonRes2.push_back( new SpinFactor("SF", SF_4Body::ONE, 0,3,1,2));   

  //////////////////////////////////////////////////////////////////////////////////////////////////

  std::vector<Lineshape*> LSK1P2Kstar1430;
  LSK1P2Kstar1430.push_back( new Lineshape("K1(1270)+", K11270M, K11270W, 1, M_23_1, LS::BW, FF::BL2) );    
  LSK1P2Kstar1430.push_back( new Lineshape("Kstar(1430)", K1430M, K1430W, 0, M_23, LS::BW, FF::BL2) ); 

  std::vector<Lineshape*> LSK1M2Kstar1430; 
  LSK1M2Kstar1430.push_back( new Lineshape("K1(1270)-", K11270M, K11270W, 1, M_14_2, LS::BW, FF::BL2) ); 
  LSK1M2Kstar1430.push_back( new Lineshape("Kstar(1430)", K1430M, K1430W,0, M_14, LS::BW, FF::BL2) );  

  std::vector<Lineshape*> LSK1P2Kstar;
  LSK1P2Kstar.push_back( new Lineshape("K1(1270)+", K11270M, K11270W, 0, M_23_1, LS::BW, FF::BL2) );    
  LSK1P2Kstar.push_back( new Lineshape("Kstar(892)", Kstar892M, Kstar892W, 1, M_23, LS::BW, FF::BL2) ); 

  std::vector<Lineshape*> LSK1M2Kstar; 
  LSK1M2Kstar.push_back( new Lineshape("K1(1270)-", K11270M, K11270W, 0, M_14_2, LS::BW, FF::BL2) ); 
  LSK1M2Kstar.push_back( new Lineshape("Kstar(892)", Kstar892M, Kstar892W,1, M_14, LS::BW, FF::BL2) );  

  std::vector<Lineshape*> LSK1P2Rho;
  LSK1P2Rho.push_back( new Lineshape("K1(1270)+", K11270M, K11270W, 0, M_12_3, LS::BW, FF::BL2) );
  LSK1P2Rho.push_back( new Lineshape("Rho(770)", RhoMass, RhoWidth, 1, M_12, LS::BW, FF::BL2) );   

  std::vector<Lineshape*> LSK1M2Rho; 
  LSK1M2Rho.push_back( new Lineshape("K1(1270)-", K11270M, K11270W, 0, M_12_4, LS::BW, FF::BL2) ); 
  LSK1M2Rho.push_back( new Lineshape("Rho(770)", RhoMass, RhoWidth, 1, M_12, LS::BW, FF::BL2) ); 
 
  std::vector<Lineshape*> LSKstarP2Kstar;
  LSKstarP2Kstar.push_back( new Lineshape("Kstar1410", Kstar1410M, Kstar1410W, 1, M_23_1, LS::BW, FF::BL2) ); 
  LSKstarP2Kstar.push_back( new Lineshape("Kstar892", Kstar892M, Kstar892W, 1, M_23, LS::BW, FF::BL2) );  

  std::vector<Lineshape*> LSKstarM2Kstar; 
  LSKstarM2Kstar.push_back( new Lineshape("Kstar1410", Kstar1410M, Kstar1410W, 1, M_14_2, LS::BW, FF::BL2) ); 
  LSKstarM2Kstar.push_back( new Lineshape("Kstar(892)", Kstar892M, Kstar892W, 1, M_14, LS::BW, FF::BL2) ); 
   
  std::vector<Lineshape*> LSKstarKstarbarS; 
  LSKstarKstarbarS.push_back( new Lineshape("Kstar(892) ", Kstar892M, Kstar892W, 1, M_23, LS::BW, FF::BL2) ); 
  LSKstarKstarbarS.push_back( new Lineshape("Kstarbar(892)", Kstar892M, Kstar892W, 1, M_14, LS::BW, FF::BL2) ); 
  
  std::vector<Lineshape*> LSKstarKstarbarP; 
  LSKstarKstarbarP.push_back( new Lineshape("Kstar(892) ", Kstar892M, Kstar892W, 1, M_23, LS::BW, FF::BL2) ); 
  LSKstarKstarbarP.push_back( new Lineshape("Kstarbar(892)", Kstar892M, Kstar892W, 1, M_14, LS::BW, FF::BL2) ); 
  
  std::vector<Lineshape*> LSKstarKstarbarD; 
  LSKstarKstarbarD.push_back( new Lineshape("Kstar(892)", Kstar892M, Kstar892W, 1, M_23, LS::BW, FF::BL2) ); 
  LSKstarKstarbarD.push_back( new Lineshape("Kstarbar(892)", Kstar892M, Kstar892W, 1, M_14, LS::BW, FF::BL2) ); 
  
  std::vector<Lineshape*> LSPhiRhoS; 
  LSPhiRhoS.push_back( new Lineshape("Phi(1020) ", phi1020M, phi1020W, 1, M_34, LS::BW, FF::BL2) ); 
  LSPhiRhoS.push_back( new Lineshape("Rho(770)", RhoMass, RhoWidth, 1, M_12, LS::BW, FF::BL2) ); 

  std::vector<Lineshape*> LSPhiRhoP; 
  LSPhiRhoP.push_back( new Lineshape("Phi(1020) ", phi1020M, phi1020W, 1, M_34, LS::BW, FF::BL2) ); 
  LSPhiRhoP.push_back( new Lineshape("Rho(770) ", RhoMass, RhoWidth, 1, M_12, LS::BW, FF::BL2) ); 


  std::vector<Lineshape*> LSPhiRhoD; 
  LSPhiRhoD.push_back( new Lineshape("Phi ", phi1020M, phi1020W, 1, M_34, LS::BW, FF::BL2) ); 
  LSPhiRhoD.push_back( new Lineshape("Rho ", RhoMass, RhoWidth, 1, M_12, LS::BW, FF::BL2) ); 

  std::vector<Lineshape*> LSPhiFZero; 
  LSPhiFZero.push_back( new Lineshape("Phi ", phi1020M, phi1020W, 1, M_34, LS::BW, FF::BL2) ); 
  LSPhiFZero.push_back( new Lineshape("Rho ", FZeroMass, FZeroWidth, 0, M_12, LS::BW, FF::BL2) ); 


 // std::vector<Lineshape*> LSPhipipi; 
 // LSPhipipi.push_back( new Lineshape("Phi ",phi1020M,phi1020W, 1, M_34, LS::BW, FF::BL2) ); 
 // LSPhipipi.push_back( new Lineshape("pipi ", new Variable("NR5", 0.0), new Variable("NR6",0.0), 1, M_12, LS::BW, FF::BL2) ); 

  std::vector<Lineshape*> LSNonRes1; 
  LSNonRes1.push_back( new Lineshape("NonRes1 ", new Variable("NR1", 0.0), new Variable("NR2", 0.0),1, M_34_2, LS::nonRes, FF::BL2) ); 

  std::vector<Lineshape*> LSNonRes2; 
  LSNonRes2.push_back( new Lineshape("NonRes2 ", new Variable("NR3", 0.0), new Variable("NR4", 0.0),1, M_34_2, LS::nonRes, FF::BL2) ); 
 //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////




// the very last parameter means that we have two permutations. so the first half of the Lineshapes 
  // and the first half of the spinfactors are amplitude 1, rest is amplitude 2
  // This means that it is important for symmetrized amplitueds that the spinfactors and lineshapes are in the "right" order
  
  //RS Model
   Amplitude* AMP_K1P2Kstar1430       = new Amplitude( "K1P2Kstar1430",   new Variable("K1P2Kstar1430_real", 1 ),     new Variable("K1P2Kstar1430_imag",  0 ), LSK1P2Kstar1430, SFK1P2Kstar1430, 1);

  Amplitude* AMP_K1M2Kstar1430       = new Amplitude( "K1M2Kstar1430",   new Variable("K1M2Kstar1430_real", 1 ),   new Variable("K1M2Kstar1430_imag", 0), LSK1M2Kstar1430, SFK1M2Kstar1430 , 1);

 //Amplitude* AMP_K1P2Kstar       = new Amplitude( "K1P2Kstar",   new Variable("K1P2Kstar_real", 1),     new Variable("K1P2Kstar_imag", 0), LSK1P2Kstar, SFK1P2Kstar, 1);

  //Amplitude* AMP_K1M2Kstar       = new Amplitude( "K1M2Kstar",   new Variable("K1M2Kstar_real", 1 ),   new Variable("K1M2Kstar_imag", 0 ), LSK1M2Kstar, SFK1M2Kstar , 1);

  //Amplitude* AMP_K1P2Rho       = new Amplitude( "K1P2Rho",   new Variable("K1P2Rho_real", 1 ), new Variable("K1P2Rho_imag", 0), LSK1P2Rho, SFK1P2Rho, 1);

  //Amplitude* AMP_K1M2Rho      = new Amplitude( "K1M2Rho",    new Variable("K1M2Rho_real", 1 ),  new Variable("K1M2Rho_imag", 0 ), LSK1M2Rho, SFK1M2Rho, 1);
  
  //Amplitude* AMP_KstarP2Kstar         = new Amplitude( "KstarP2Kstar",      new Variable("KstarP2Kstar_real",1),  new Variable("KstarP2Kstar_imag", 0 ), LSKstarP2Kstar, SFKstarP2Kstar, 1);
 
 //Amplitude* AMP_KstarM2Kstar     = new Amplitude( "KstarM2Kstar", new Variable("KstarM2Kstar_real", 1 ),   new Variable("KstarM2Kstar_imag", 0), LSKstarM2Kstar  , SFKstarM2Kstar , 1);
   
 //Amplitude* AMP_KstarKstarbarS          = new Amplitude( "KstarKstarbarS",      new Variable("KstarKstarbarS_real", 1 ),  new Variable("KstarKstarbarS_imag", 0), LSKstarKstarbarS  , SFKstarKstarS , 1);
 

 //Amplitude* AMP_KstarKstarbarP          = new Amplitude( "KstarKstarbarP",      new Variable("KstarKstarbarP_real", 1),  new Variable("KstarKstarbarP_imag", 0), LSKstarKstarbarP  , SFKstarKstarP , 1);
 

 //Amplitude* AMP_KstarKstarbarD          = new Amplitude( "KstarKstarbarD",      new Variable("KstarKstarbarD_real", 1),  new Variable("KstarKstarbarD_imag", 0  ), LSKstarKstarbarD  , SFKstarKstarD , 1);
  
 //Amplitude* AMP_PhiRhoS       = new Amplitude( "PhiRhoS",   new Variable("PhiRhoS_real",   1),  new Variable("PhiRhoS_imag", 0), LSPhiRhoS  , SFPhiRhoS , 1);


 //Amplitude* AMP_PhiRhoP       = new Amplitude( "PhiRhoP",   new Variable("PhiRhoP_real", 1 ),  new Variable("PhiRhoP_imag", 0 ), LSPhiRhoP  , SFPhiRhoP , 1);

  //Amplitude* AMP_PhiRhoD        = new Amplitude( "PhiRhoD",    new Variable("PhiRhoD", 1  ),   new Variable("PhiRhoD_imag", 0  ), LSPhiRhoD  , SFPhiRhoD , 1);
 
  Amplitude* AMP_PhiFZero        = new Amplitude( "PhiFZero",    new Variable("PhiFZero",1  ),   new Variable("PhiFZero_imag", 0 ), LSPhiRhoD  , SFPhiRhoD , 1);
 
 Amplitude* AMP_NonRes1      = new Amplitude( "NonRes1",  new Variable("Nonres1_real", 0.179343),  new Variable("NonRes1_imag", 0.0  ), LSNonRes1  , SFNonRes1 , 1);

  Amplitude* AMP_NonRes2     = new Amplitude( "NonRes2", new Variable("NonRes2_real", 1.79348e-01 ),  new Variable("NonRes2_imag", 1.34666e1  ), LSNonRes2  , SFNonRes2 , 1);

/////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////


  //DKKPP_DI->amplitudes.push_back(AMP_K1P2Kstar1430);
  //DKKPP_DI->amplitudes.push_back(AMP_K1M2Kstar1430);
  //DKKPP_DI->amplitudes.push_back(AMP_K1P2Kstar);
  //DKKPP_DI->amplitudes.push_back(AMP_K1M2Kstar);
  //DKKPP_DI->amplitudes.push_back(AMP_K1P2Rho);
  //DKKPP_DI->amplitudes.push_back(AMP_K1M2Rho);
  //DKKPP_DI->amplitudes.push_back(AMP_KstarP2Kstar);
  //DKKPP_DI->amplitudes.push_back(AMP_KstarM2Kstar);
  //DKKPP_DI->amplitudes.push_back(AMP_KstarKstarbarS);
  //DKKPP_DI->amplitudes.push_back(AMP_KstarKstarbarP);
  //DKKPP_DI->amplitudes.push_back(AMP_KstarKstarbarD);
  //DKKPP_DI->amplitudes.push_back(AMP_PhiRhoS);
  //DKKPP_DI->amplitudes.push_back(AMP_PhiRhoP);
  //DKKPP_DI->amplitudes.push_back(AMP_PhiRhoD);
  DKKPP_DI->amplitudes.push_back(AMP_PhiFZero); 
  //DKKPP_DI->amplitudes.push_back(AMP_NonRes1);
  //DKKPP_DI->amplitudes.push_back(AMP_NonRes2);
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////i////////////////////////////////////////////////////////////////////////////


  DKKPP_DI->_tau = new Variable("tau", 0.4101);
  DKKPP_DI->_xmixing = new Variable("xmixing", 0.0049);
  DKKPP_DI->_ymixing = new Variable("ymixing", 0.0061);
  // DK3P_DI->_xmixing = new Variable("xmixing", 0, 0.00001, -.15, .15);
  // DK3P_DI->_ymixing = new Variable("ymixing", 0, 0.00001, -.15, .15);
  DKKPP_DI->_SqWStoRSrate = new Variable("SqWStoRSrate", 1.0/sqrt(300.0));  


  Variable* m12 = new Variable("m12", 0, 3);
  Variable* m34 = new Variable("m34", 0, 3); 
  Variable* cos12 = new Variable("cos12", -1, 1);
  Variable* cos34 = new Variable("cos34", -1, 1);
  Variable* phi = new Variable("phi", -3.5, 3.5);
  Variable* eventNumber = new Variable("eventNumber", 0, INT_MAX);
  //Variable* dtime = new Variable("dtime", 0, 10);
  //Variable* sigmat = new Variable("sigmat",-3,3);
  Variable* constantOne = new Variable("constantOne", 1); 
  Variable* constantZero = new Variable("constantZero", 0);
 

  std::vector<Variable*> vars;
  vars.push_back(m12);
  vars.push_back(m34);
  vars.push_back(cos12);
  vars.push_back(cos34);
  vars.push_back(phi);
  vars.push_back(eventNumber); 
  //vars.push_back(dtime); 
  //vars.push_back(sigmat); 
  UnbinnedDataSet currData(vars); 

 
  //DKKPP_DI->_xmixing->value = strtof(argv[5], NULL);
  //DKKPP_DI->_ymixing->value = strtof(argv[6], NULL);

  vector<Variable*> observables;
  vector<Variable*> coefficients; 
  vector<Variable*> offsets;

  observables.push_back(m12);
  observables.push_back(m34);
  observables.push_back(cos12);
  observables.push_back(cos34);
  observables.push_back(phi);
  observables.push_back(eventNumber);
  //observables.push_back(dtime);
 // observables.push_back(sigmat);
  offsets.push_back(constantZero);
  offsets.push_back(constantZero);
  coefficients.push_back(constantOne); 
  fprintf(stderr, "I'm here zero"); 
  TruthResolution* dat = new TruthResolution();
  PolynomialPdf* eff = new PolynomialPdf("constantEff", observables, coefficients, offsets, 0);
  DPPdf* dp = new DPPdf("test_TD", observables, DKKPP_DI, eff, 1);
 
 // dp->setGenDecayTimeLimit(0,3.5); // this corresponds to rougly 97% of the exponential. So this should be ok. And speeds up Generation significantly compared to [0,5]  

  fprintf(stderr,"I'm here one\n"); 

  TFile *file = new TFile( argv[4] , "RECREATE");
  TTree *tree = new TTree("events", "events");

  double tm12_2,tm34_2,tc12_2,tc34_2,tphi_2,tdtime_2;

  tree->Branch("m12",          &tm12_2,         "m12/D");
  tree->Branch("m34",          &tm34_2,         "m34/D");
  tree->Branch("c12",          &tc12_2,         "c12/D");
  tree->Branch("c34",          &tc34_2,         "c34/D");
  tree->Branch("phi",          &tphi_2,         "phi/D");

  fprintf(stderr, "I'm here two\n"); 

  //mcbooster::FlagAcceptReject(0,0);
  int generatedEvents = 0;
  int RunNum = 0;
  fprintf(stderr, "I'm here three\n");
  int BatchSize = strtoul(argv[1], NULL,0);
  fprintf(stderr, "I'm here three\n"); 
  unsigned int offi = strtoul(argv[3], NULL,0);
  unsigned int genEvts =strtoul(argv[2], NULL,0);

  double wmax = 0;
  //mcbooster::FlagAcceptReject FlagIt =1;// mcbooster::FlagAcceptReject(0.1,5);

  fprintf(stderr, "I'm here"); 
  while(generatedEvents < genEvts )
  {
    fprintf(stderr,"I'm here"); 
    unsigned int keptEvts = 0;
    dp->setGenerationOffset(offi);
    auto tuple = dp->GenerateSig(BatchSize);
    fprintf(stderr,"after gen\n");
    auto particles = std::get<0>(tuple);
    auto variables = std::get<1>(tuple);
    auto weights = std::get<2>(tuple);
    auto flags = std::get<3>(tuple);
    // int accepted = thrust::count_if(flags.begin(), flags.end(), thrust::identity<bool>());
    ++RunNum;
    // generatedEvents += accepted;
    fprintf(stderr,"after gen\n");

    for (int i = 0; i < weights.size(); ++i)
    {
      if (wmax<weights[i]) wmax = weights[i];
      if (generatedEvents < genEvts && flags[i]==1){
        ++generatedEvents;
        ++keptEvts;
        // printf("PF %i: %s %.5g %.5g %.5g %.5g %.5g %.5g\n",i, (bool)flags[i] ? "true" : "false", weights[i], (*(variables[0]))[i], (*(variables[1]))[i], (*(variables[2]))[i], (*(variables[3]))[i], (*(variables[4]))[i]);
        tm12_2 = (*(variables[0]))[i];
        tm34_2 = (*(variables[1]))[i];
        tc12_2 = (*(variables[2]))[i];
        tc34_2 = (*(variables[3]))[i];
        tphi_2 = (*(variables[4]))[i];
        tree->Fill();
        // printf("Buffer %i: %.5g %.5g %.5g %.5g %.5g %.5g \n",i, (*myweights)[i],(*Buffer_m12)[i], (*Buffer_m34)[i], (*Buffer_c12)[i], (*Buffer_c34)[i], (*Buffer_phi)[i], (*Buffer_dt)[i]);
      }
    }
    fprintf(stderr,"Run # %i: x=%.6g y=%.6g Using accept-reject method leaves you with %i out of %i events.  %.4g %% of Total offset: %u\n",RunNum, DKKPP_DI->_xmixing->value, DKKPP_DI->_ymixing->value, keptEvts, BatchSize, generatedEvents*100.0/genEvts, offi);
    offi += BatchSize;
    delete variables[0];
    delete variables[1];
    delete variables[2];
    delete variables[3];
    delete variables[4];

    delete particles[0];
    delete particles[1];
    delete particles[2];
    delete particles[3];
  }
  // printf("start\n");
  // int i = 0;
  // printf("Buffer %i: %.5g %.5g %.5g %.5g %.5g %.5g \n",i, (*myweights)[i],(*Buffer_m12)[i], (*Buffer_m34)[i], (*Buffer_c12)[i], (*Buffer_c34)[i], (*Buffer_phi)[i], (*Buffer_dt)[i]);

  // printf("start2\n");
  std::ofstream out;
  string outname ="Max_observed_weights.txt";
  out.open(outname.c_str(), std::ios::app);
  out.precision(10);

  out << wmax <<endl;

  tree->Write();
  file->Close();
  // printf("overall wmax %f, keept %u evts, reweight ratio %.5g\n",wmax, keptEvts, (double)keptEvts/genEvts );
  printf("%i\n",offi);
  //std::cout<<amps[2]<<std::endl;
  return 0; 

}
