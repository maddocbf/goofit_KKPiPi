#include "hip/hip_runtime.h"
//THIS PROGRAM GENERATES MONTECARLO DATA GIVEN AN AMPLITUDE MODEL


//ROOT
#include <TFile.h>
#include <TTree.h>
#include <iostream>
#include <string>

// GooFit stuff
#include "goofit/Variable.h" 
#include "goofit/PDFs/PolynomialPdf.h" 
#include "goofit/UnbinnedDataSet.h"
#include "goofit/PDFs/DP4Pdf.h"
#include "goofit/PDFs/TruthResolution_Aux.h" 
#include "goofit/PDFs/Tddp4Pdf.h"
#include <thrust/count.h>
#include <fstream>
#include <iomanip>
#include <numeric>
#include <algorithm>
#include <random>
#include <ctime>
#include <functional>
#include <mcbooster/functors/FlagAcceptReject.h>

using namespace std;

// Constants used in more than one PDF component. 
const fptype _mD0 = 1.8645; 
const fptype piPlusMass = 0.13957018;
const fptype KmMass = .493677;
int main (int argc, char** argv) {

  // hipSetDevice(0);

  DecayInfo_DP* DK3P_DI = new DecayInfo_DP();
  DK3P_DI->meson_radius =5;
  DK3P_DI->particle_masses.push_back(_mD0);
  DK3P_DI->particle_masses.push_back(piPlusMass);
  DK3P_DI->particle_masses.push_back(piPlusMass);
  DK3P_DI->particle_masses.push_back(KmMass);
  DK3P_DI->particle_masses.push_back(piPlusMass);
 
  Variable* RhoMass  =  new Variable("rho_mass", 0.77526);
  Variable* RhoWidth =  new Variable("rho_width", 0.1478); 
  Variable* K892M   =   new Variable("K892M", 0.89581);
  Variable* K892W   =   new Variable("K892W", 0.0474); 
  Variable* f600M  =    new Variable("f600M", 0.519);
  Variable* f600W  =    new Variable("f600W", 0.454); 
  Variable* a1M  =      new Variable("a1M", 1.237);
  Variable* a1W  =      new Variable("a1W", 0.526); 
  Variable* K1_1270M  = new Variable("K1_1270M", 1.28241);
  Variable* K1_1270W  = new Variable("K1_1270W", 0.06596); 
  Variable* K0_1430M  = new Variable("K0_1430M", 1.425);
  Variable* K0_1430W  = new Variable("K0_1430W", 0.27);

  Variable* K1410M    = new Variable("K1410M", 1.414);
  Variable* K1410W    = new Variable("K1410W", 0.232); 

  Variable* rho1450M  = new Variable("rho1450M", 1.465);
  Variable* rho1450W  = new Variable("rho1450W", 0.400); 

  Variable* K1460M    = new Variable("K1460M", 1.351);
  Variable* K1460W    = new Variable("K1460W", 0.281); 

  Variable* f0_1370M  = new Variable("f0_1370M", 1.350);
  Variable* f0_1370W  = new Variable("f0_1370W", 0.35); 

  Variable* K1_1400M  = new Variable("K1_1400M", 1.403);
  Variable* K1_1400W  = new Variable("K1_1400W", 0.174); 

  Variable* K2_1430M  = new Variable("K2_1430M", 1.4256);
  Variable* K2_1430W  = new Variable("K2_1430W", 0.0985); 
  
  std::vector<Variable*> LassVars;
  LassVars.push_back( new Variable("lass_a",2.07) );
  LassVars.push_back( new Variable("lass_r",3.32) );
  LassVars.push_back( new Variable("lass_pf",0.0) );
  LassVars.push_back( new Variable("lass_pr",0.0) );
  LassVars.push_back( new Variable("lass_F",1.0) );

 //Spin factors: we have two due to the bose symmetrization of the two pi+
  std::vector<SpinFactor*> SF_K892_rho770_S;
  SF_K892_rho770_S.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_S, 0, 1, 2, 3) );
  SF_K892_rho770_S.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_S, 3, 1, 2, 0) );
  //Lineshapes, also for both pi+ configurations
  std::vector<Lineshape*> LS_K892_rho770_S;
  LS_K892_rho770_S.push_back( new Lineshape("K*(892)bar", K892M, K892W, 1, M_34, LS::BW, FF::BL2) );
  LS_K892_rho770_S.push_back( new Lineshape("rho(770)", RhoMass, RhoWidth, 1, M_12, LS::BW, FF::BL2) );
  LS_K892_rho770_S.push_back( new Lineshape("K*(892)bar", K892M, K892W, 1, M_13, LS::BW, FF::BL2) );
  LS_K892_rho770_S.push_back( new Lineshape("rho(770)", RhoMass, RhoWidth, 1, M_24, LS::BW, FF::BL2) );

  std::vector<SpinFactor*> SF_K892_rho770_P;
  SF_K892_rho770_P.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_P, 0, 1, 2, 3) );
  SF_K892_rho770_P.push_back( new SpinFactor("SF", SF_4Body::FF_12_34_L1, 0, 1, 2, 3) );
  SF_K892_rho770_P.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_P, 3, 1, 2, 0) );
  SF_K892_rho770_P.push_back( new SpinFactor("SF", SF_4Body::FF_12_34_L1, 3, 1, 2, 0) );
  std::vector<Lineshape*> LS_K892_rho770_P;
  LS_K892_rho770_P.push_back( new Lineshape("rho(770)", RhoMass, RhoWidth, 1, M_12, LS::BW, FF::BL2) );
  LS_K892_rho770_P.push_back( new Lineshape("K*(892)bar", K892M, K892W, 1, M_34, LS::BW, FF::BL2) );
  LS_K892_rho770_P.push_back( new Lineshape("rho(770)", RhoMass, RhoWidth, 1, M_24, LS::BW, FF::BL2) );
  LS_K892_rho770_P.push_back( new Lineshape("K*(892)bar", K892M, K892W, 1, M_13, LS::BW, FF::BL2) );

  std::vector<SpinFactor*> SF_K892_rho770_D;
  SF_K892_rho770_D.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_D, 0, 1, 2, 3) );
  SF_K892_rho770_D.push_back( new SpinFactor("SF", SF_4Body::FF_12_34_L2, 0, 1, 2, 3) );
  SF_K892_rho770_D.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_D, 3, 1, 2, 0) );
  SF_K892_rho770_D.push_back( new SpinFactor("SF", SF_4Body::FF_12_34_L2, 3, 1, 2, 0) );
  std::vector<Lineshape*> LS_K892_rho770_D;
  LS_K892_rho770_D.push_back( new Lineshape("rho(770)", RhoMass, RhoWidth, 1, M_12, LS::BW, FF::BL2) );
  LS_K892_rho770_D.push_back( new Lineshape("K*(892)bar", K892M, K892W, 1, M_34, LS::BW, FF::BL2) );
  LS_K892_rho770_D.push_back( new Lineshape("rho(770)", RhoMass, RhoWidth, 1, M_24, LS::BW, FF::BL2) );
  LS_K892_rho770_D.push_back( new Lineshape("K*(892)bar", K892M, K892W, 1, M_13, LS::BW, FF::BL2) );

  std::vector<SpinFactor*> SF_K1410_rho770_S;
  SF_K1410_rho770_S.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_S, 0, 1, 2, 3) );
  SF_K1410_rho770_S.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_S, 3, 1, 2, 0) );
  std::vector<Lineshape*> LS_K1410_rho770_S;
  LS_K1410_rho770_S.push_back( new Lineshape("rho(770)", RhoMass, RhoWidth, 1, M_12, LS::BW, FF::BL2) );
  LS_K1410_rho770_S.push_back( new Lineshape("K*(1410)", K1410M, K1410W, 1, M_34, LS::BW, FF::BL2) );
  LS_K1410_rho770_S.push_back( new Lineshape("rho(770)", RhoMass, RhoWidth, 1, M_24, LS::BW, FF::BL2) );
  LS_K1410_rho770_S.push_back( new Lineshape("K*(1410)", K1410M, K1410W, 1, M_13, LS::BW, FF::BL2) );


  std::vector<SpinFactor*> SF_K1410_rho770_P;
  SF_K1410_rho770_P.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_P, 0, 1, 2, 3) );
  SF_K1410_rho770_P.push_back( new SpinFactor("SF", SF_4Body::FF_12_34_L1, 0, 1, 2, 3) );
  SF_K1410_rho770_P.push_back( new SpinFactor("SF", SF_4Body::DtoV1V2_V1toP1P2_V2toP3P4_P, 3, 1, 2, 0) );
  SF_K1410_rho770_P.push_back( new SpinFactor("SF", SF_4Body::FF_12_34_L1, 3, 1, 2, 0) );
  std::vector<Lineshape*> LS_K1410_rho770_P;
  LS_K1410_rho770_P.push_back( new Lineshape("rho(770)", RhoMass, RhoWidth, 1, M_12, LS::BW, FF::BL2) );
  LS_K1410_rho770_P.push_back( new Lineshape("K*(1410)", K1410M, K1410W, 1, M_34, LS::BW, FF::BL2) );
  LS_K1410_rho770_P.push_back( new Lineshape("rho(770)", RhoMass, RhoWidth, 1, M_24, LS::BW, FF::BL2) );
  LS_K1410_rho770_P.push_back( new Lineshape("K*(1410)", K1410M, K1410W, 1, M_13, LS::BW, FF::BL2) );

  std::vector<SpinFactor*> SF_K892_f0_600;
  SF_K892_f0_600.push_back( new SpinFactor("SF", SF_4Body::DtoVS_VtoP1P2_StoP3P4, 2, 3, 0, 1) );
  SF_K892_f0_600.push_back( new SpinFactor("SF", SF_4Body::FF_12_34_L1, 2, 3, 0, 1) );
  SF_K892_f0_600.push_back( new SpinFactor("SF", SF_4Body::DtoVS_VtoP1P2_StoP3P4, 2, 0, 3, 1) );
  SF_K892_f0_600.push_back( new SpinFactor("SF", SF_4Body::FF_12_34_L1, 2, 0, 3, 1) );
  std::vector<Lineshape*> LS_K892_f0_600;
  LS_K892_f0_600.push_back( new Lineshape("K*(892)bar", K892M, K892W, 1, M_34, LS::BW, FF::BL2) );
  LS_K892_f0_600.push_back( new Lineshape("f600", f600M, f600W, 0, M_12, LS::Bugg3, FF::BL2) );
  LS_K892_f0_600.push_back( new Lineshape("K*(892)bar", K892M, K892W, 1, M_13, LS::BW, FF::BL2) );
  LS_K892_f0_600.push_back( new Lineshape("f600", f600M, f600W, 0, M_24, LS::Bugg3, FF::BL2) );

  std::vector<SpinFactor*> SF_rho1450_K0_1430;
  SF_rho1450_K0_1430.push_back( new SpinFactor("SF", SF_4Body::DtoVS_VtoP1P2_StoP3P4, 0, 1, 2, 3) );
  SF_rho1450_K0_1430.push_back( new SpinFactor("SF", SF_4Body::FF_12_34_L1, 0, 1, 2, 3) );
  SF_rho1450_K0_1430.push_back( new SpinFactor("SF", SF_4Body::DtoVS_VtoP1P2_StoP3P4, 3, 1, 2, 0) );
  SF_rho1450_K0_1430.push_back( new SpinFactor("SF", SF_4Body::FF_12_34_L1, 3, 1, 2, 0) );
  std::vector<Lineshape*> LS_rho1450_K0_1430;
  LS_rho1450_K0_1430.push_back( new Lineshape("rho(1450)", rho1450M, rho1450W, 1, M_12, LS::BW, FF::BL2) );
  LS_rho1450_K0_1430.push_back( new Lineshape("K(0)*(1430)", K0_1430M, K0_1430W, 0, M_34, LS::Lass_M3, FF::BL2, 1.5, LassVars) );
  LS_rho1450_K0_1430.push_back( new Lineshape("rho(1450)", rho1450M, rho1450W, 1, M_24, LS::BW, FF::BL2) );
  LS_rho1450_K0_1430.push_back( new Lineshape("K(0)*(1430)", K0_1430M, K0_1430W, 0, M_13, LS::Lass_M3, FF::BL2, 1.5, LassVars) );

  std::vector<SpinFactor*> SF_K1460_K892;
  SF_K1460_K892.push_back( new SpinFactor("SF", SF_4Body::DtoPP1_PtoVP2_VtoP3P4, 0, 1, 2, 3) );
  SF_K1460_K892.push_back( new SpinFactor("SF", SF_4Body::DtoPP1_PtoVP2_VtoP3P4, 3, 1, 2, 0) );
  std::vector<Lineshape*> LS_K1460_K892;
  LS_K1460_K892.push_back( new Lineshape("K1460", K1460M, K1460W, 1, M_34_2, LS::BW, FF::BL2) );
  LS_K1460_K892.push_back( new Lineshape("K*(892)bar", K892M, K892W, 1, M_34, LS::BW, FF::BL2) );
  LS_K1460_K892.push_back( new Lineshape("K1460", K1460M, K1460W, 1, M_13_2, LS::BW, FF::BL2) );
  LS_K1460_K892.push_back( new Lineshape("K*(892)bar", K892M, K892W, 1, M_13, LS::BW, FF::BL2) );

  std::vector<SpinFactor*> SF_K1460_f0_1370;
  SF_K1460_f0_1370.push_back( new SpinFactor("SF", SF_4Body::DtoPP1_PtoSP2_StoP3P4, 0, 1, 2, 3) );
  SF_K1460_f0_1370.push_back( new SpinFactor("SF", SF_4Body::DtoPP1_PtoSP2_StoP3P4, 3, 1, 2, 0) );
  std::vector<Lineshape*> LS_K1460_f0_1370;
  LS_K1460_f0_1370.push_back( new Lineshape("K1460", K1460M, K1460W, 0, M_12_3, LS::BW, FF::BL2) );
  LS_K1460_f0_1370.push_back( new Lineshape("f0_1370", f0_1370M, f0_1370W, 0, M_12, LS::BW, FF::BL2) );
  LS_K1460_f0_1370.push_back( new Lineshape("K1460", K1460M, K1460W, 0, M_24_3, LS::BW, FF::BL2) );
  LS_K1460_f0_1370.push_back( new Lineshape("f0_1370", f0_1370M, f0_1370W, 0, M_24, LS::BW, FF::BL2) );

  std::vector<SpinFactor*> SF_K1_1270_K892;
  SF_K1_1270_K892.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, 0, 1, 2, 3) );
  SF_K1_1270_K892.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1, 1, 2, 3, 0) );
  SF_K1_1270_K892.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, 3, 1, 2, 0) );
  SF_K1_1270_K892.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1, 1, 2, 0, 3) );
  std::vector<Lineshape*> LS_K1_1270_K892;
  LS_K1_1270_K892.push_back( new Lineshape("K1_1270", K1_1270M, K1_1270W, 0, M_34_2, LS::BW, FF::BL2) );
  LS_K1_1270_K892.push_back( new Lineshape("K*(892)bar", K892M, K892W, 1, M_34, LS::BW, FF::BL2) );
  LS_K1_1270_K892.push_back( new Lineshape("K1_1270", K1_1270M, K1_1270W, 0, M_13_2, LS::BW, FF::BL2) );
  LS_K1_1270_K892.push_back( new Lineshape("K*(892)bar", K892M, K892W, 1, M_13, LS::BW, FF::BL2) );

  std::vector<SpinFactor*> SF_K1_1270_rho770;
  SF_K1_1270_rho770.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, 3, 2, 0, 1) );
  SF_K1_1270_rho770.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1, 0, 1, 2, 3) );
  SF_K1_1270_rho770.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, 0, 2, 3, 1) );
  SF_K1_1270_rho770.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1, 1, 2, 3, 0) );
  std::vector<Lineshape*> LS_K1_1270_rho770;
  LS_K1_1270_rho770.push_back( new Lineshape("K1_1270", K1_1270M, K1_1270W, 0, M_12_3, LS::BW, FF::BL2) );
  LS_K1_1270_rho770.push_back( new Lineshape("rho(770)", RhoMass, RhoWidth, 1, M_12, LS::BW, FF::BL2) );
  LS_K1_1270_rho770.push_back( new Lineshape("K1_1270", K1_1270M, K1_1270W, 0, M_24_3, LS::BW, FF::BL2) );
  LS_K1_1270_rho770.push_back( new Lineshape("rho(770)", RhoMass, RhoWidth, 1, M_24, LS::BW, FF::BL2) );

  std::vector<SpinFactor*> SF_K1_1270_K0_1430;
  SF_K1_1270_K0_1430.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoSP2_StoP3P4, 0, 1, 2, 3) );
  SF_K1_1270_K0_1430.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1, 1, 2, 3, 0) );
  SF_K1_1270_K0_1430.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoSP2_StoP3P4, 3, 1, 2, 0) );
  SF_K1_1270_K0_1430.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1, 0, 1, 2, 3) );
  std::vector<Lineshape*> LS_K1_1270_K0_1430;
  LS_K1_1270_K0_1430.push_back( new Lineshape("K(1)(1270)bar", K1_1270M, K1_1270W, 1, M_34_2 , LS::BW, FF::BL2) );
  LS_K1_1270_K0_1430.push_back( new Lineshape("K(0)*(1430)", K0_1430M, K0_1430W, 0, M_34 , LS::Lass_M3, FF::BL2, 1.5, LassVars) );
  LS_K1_1270_K0_1430.push_back( new Lineshape("K(1)(1270)bar2", K1_1270M, K1_1270W, 1, M_13_2 , LS::BW, FF::BL2) );
  LS_K1_1270_K0_1430.push_back( new Lineshape("K(0)*1430)", K0_1430M, K0_1430W, 0, M_13 , LS::Lass_M3, FF::BL2, 1.5, LassVars) );

  std::vector<SpinFactor*> SF_K1_1400_K892;
  SF_K1_1400_K892.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, 0, 1, 2, 3) );
  SF_K1_1400_K892.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1, 1, 2, 3, 0) );
  SF_K1_1400_K892.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, 3, 1, 2, 0) );
  SF_K1_1400_K892.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1, 1, 2, 0, 3) );
  std::vector<Lineshape*> LS_K1_1400_K892;
  LS_K1_1400_K892.push_back( new Lineshape("K1_1400", K1_1400M, K1_1400W, 0, M_34_2, LS::BW, FF::BL2) );
  LS_K1_1400_K892.push_back( new Lineshape("K*(892)bar", K892M, K892W, 1, M_34, LS::BW, FF::BL2) );
  LS_K1_1400_K892.push_back( new Lineshape("K1_1400", K1_1400M, K1_1400W, 0, M_13_2, LS::BW, FF::BL2) );
  LS_K1_1400_K892.push_back( new Lineshape("K*(892)bar", K892M, K892W, 1, M_13, LS::BW, FF::BL2) );

  std::vector<SpinFactor*> SF_K2_1430_K892;
  SF_K2_1430_K892.push_back( new SpinFactor("SF", SF_4Body::DtoTP1_TtoVP2_VtoP3P4, 0, 1, 2, 3) );
  SF_K2_1430_K892.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L2, 1, 2, 3, 0) );
  SF_K2_1430_K892.push_back( new SpinFactor("SF", SF_4Body::DtoTP1_TtoVP2_VtoP3P4, 3, 1, 2, 0) );
  SF_K2_1430_K892.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L2, 1, 2, 0, 3) );
  std::vector<Lineshape*> LS_K2_1430_K892;
  LS_K2_1430_K892.push_back( new Lineshape("K2_1430", K2_1430M, K2_1430W, 2, M_34_2, LS::BW, FF::BL2) );
  LS_K2_1430_K892.push_back( new Lineshape("K*(892)bar", K892M, K892W, 1, M_34, LS::BW, FF::BL2) );
  LS_K2_1430_K892.push_back( new Lineshape("K2_1430", K2_1430M, K2_1430W, 2, M_13_2, LS::BW, FF::BL2) );
  LS_K2_1430_K892.push_back( new Lineshape("K*(892)bar", K892M, K892W, 1, M_13, LS::BW, FF::BL2) );

  std::vector<SpinFactor*> SF_K2_1430_rho770;
  SF_K2_1430_rho770.push_back( new SpinFactor("SF", SF_4Body::DtoTP1_TtoVP2_VtoP3P4, 3, 2, 0, 1) );
  SF_K2_1430_rho770.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L2, 0, 1, 2, 3) );
  SF_K2_1430_rho770.push_back( new SpinFactor("SF", SF_4Body::DtoTP1_TtoVP2_VtoP3P4, 0, 2, 3, 1) );
  SF_K2_1430_rho770.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L2, 3, 1, 2, 0) );
  std::vector<Lineshape*> LS_K2_1430_rho770;
  LS_K2_1430_rho770.push_back( new Lineshape("K2_1430", K2_1430M, K2_1430W, 2, M_12_3, LS::BW, FF::BL2) );
  LS_K2_1430_rho770.push_back( new Lineshape("rho(770)", RhoMass, RhoWidth, 1, M_12, LS::BW, FF::BL2) );
  LS_K2_1430_rho770.push_back( new Lineshape("K2_1430", K2_1430M, K2_1430W, 2, M_24_3, LS::BW, FF::BL2) );
  LS_K2_1430_rho770.push_back( new Lineshape("rho(770)", RhoMass, RhoWidth, 1, M_24, LS::BW, FF::BL2) );

  std::vector<SpinFactor*> SF_a1_f0_600;
  SF_a1_f0_600.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoSP2_StoP3P4, 2, 3, 0, 1) );
  SF_a1_f0_600.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1, 0, 1, 3, 2) );
  SF_a1_f0_600.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoSP2_StoP3P4, 2, 0, 3, 1) );
  SF_a1_f0_600.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1, 0, 1, 3, 2) );
  std::vector<Lineshape*> LS_a1_f0_600;
  LS_a1_f0_600.push_back( new Lineshape("a(1)(1260)+", a1M, a1W, 1, M_12_4, LS::BW, FF::BL2, 5.71) );
  LS_a1_f0_600.push_back( new Lineshape("f600", f600M, f600W, 0, M_12, LS::Bugg3, FF::BL2) );
  LS_a1_f0_600.push_back( new Lineshape("a(1)(1260)+", a1M, a1W, 1, M_24_1, LS::BW, FF::BL2, 5.71) );
  LS_a1_f0_600.push_back( new Lineshape("f600", f600M, f600W, 0, M_24, LS::Bugg3, FF::BL2) );

  std::vector<SpinFactor*> SF_a1_rho770;
  SF_a1_rho770.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, 2, 3, 0, 1) );
  SF_a1_rho770.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1, 0, 1, 3, 2) );
  SF_a1_rho770.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2_VtoP3P4, 2, 0, 3, 1) );
  SF_a1_rho770.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1, 0, 1, 3, 2) );
  std::vector<Lineshape*> LS_a1_rho770;
  LS_a1_rho770.push_back( new Lineshape("a(1)(1260)+", a1M, a1W, 0, M_12_4, LS::BW, FF::BL2, 5.71) );
  LS_a1_rho770.push_back( new Lineshape("rho(770)", RhoMass, RhoWidth, 1, M_12, LS::BW, FF::BL2) );
  LS_a1_rho770.push_back( new Lineshape("a(1)(1260)+", a1M, a1W, 0, M_24_1, LS::BW, FF::BL2, 5.71) );
  LS_a1_rho770.push_back( new Lineshape("rho(770)", RhoMass, RhoWidth, 1, M_24, LS::BW, FF::BL2) );

  std::vector<SpinFactor*> SF_a1_rho770_D;
  SF_a1_rho770_D.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2Dwave_VtoP3P4, 2, 3, 0, 1) );
  SF_a1_rho770_D.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1, 0, 1, 3, 2) );
  SF_a1_rho770_D.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2Dwave_VtoP3P4, 2, 0, 3, 1) );
  SF_a1_rho770_D.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1, 0, 1, 3, 2) );
  std::vector<Lineshape*> LS_a1_rho770_D;
  LS_a1_rho770_D.push_back( new Lineshape("a(1)(1260)+", a1M, a1W, 2, M_12_4, LS::BW, FF::BL2, 5.71) );
  LS_a1_rho770_D.push_back( new Lineshape("rho(770)", RhoMass, RhoWidth, 1, M_12, LS::BW, FF::BL2) );
  LS_a1_rho770_D.push_back( new Lineshape("a(1)(1260)+", a1M, a1W, 2, M_24_1, LS::BW, FF::BL2, 5.71) );
  LS_a1_rho770_D.push_back( new Lineshape("rho(770)", RhoMass, RhoWidth, 1, M_24, LS::BW, FF::BL2) );

  std::vector<SpinFactor*> SF_nonRes;
  SF_nonRes.push_back( new SpinFactor("SF", SF_4Body::ONE, 2, 3, 0, 1) );
  SF_nonRes.push_back( new SpinFactor("SF", SF_4Body::ONE, 2, 0, 3, 1) );
  std::vector<Lineshape*> LS_nonRes;
  LS_nonRes.push_back( new Lineshape("nonRes", a1M,     a1W,          0, M_12, LS::ONE, FF::BL2) );
  LS_nonRes.push_back( new Lineshape("nonRes", RhoMass, RhoWidth,     0, M_34, LS::ONE, FF::BL2) );
  LS_nonRes.push_back( new Lineshape("nonRes", a1M,     a1W,          0, M_12, LS::ONE, FF::BL2) );
  LS_nonRes.push_back( new Lineshape("nonRes", RhoMass, RhoWidth,     0, M_34, LS::ONE, FF::BL2) );

  std::vector<SpinFactor*> SF_NonResA_K892;
  SF_NonResA_K892.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2Dwave_VtoP3P4, 0, 1, 2, 3) );
  SF_NonResA_K892.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1, 1, 2, 3, 0) );
  SF_NonResA_K892.push_back( new SpinFactor("SF", SF_4Body::DtoAP1_AtoVP2Dwave_VtoP3P4, 3, 1, 2, 0) );
  SF_NonResA_K892.push_back( new SpinFactor("SF", SF_4Body::FF_123_4_L1, 1, 2, 0, 3) );
  std::vector<Lineshape*> LS_NonResA_K892;
  LS_NonResA_K892.push_back( new Lineshape("K1_1400", new Variable("NR1",0.0), new Variable("NR2",0.0), 2, M_34_2, LS::nonRes, FF::BL2) );
  LS_NonResA_K892.push_back( new Lineshape("K*(892)bar", K892M, K892W, 1, M_34, LS::BW, FF::BL2) );
  LS_NonResA_K892.push_back( new Lineshape("K1_1400", new Variable("NR3",0.0), new Variable("NR4",0.0), 2, M_13_2, LS::nonRes, FF::BL2) );
  LS_NonResA_K892.push_back( new Lineshape("K*(892)bar", K892M, K892W, 1, M_13, LS::BW, FF::BL2) );

  // the very last parameter means that we have two permutations. so the first half of the Lineshapes 
  // and the first half of the spinfactors are amplitude 1, rest is amplitude 2
  // This means that it is important for symmetrized amplitueds that the spinfactors and lineshapes are in the "right" order
  
  //RS Model
  Amplitude* amp_K892_rho770_S       = new Amplitude( "K892_rho770_S",   new Variable("K892_rho770_S_real",   1.0),     new Variable("K892_rho770_S_imag", 0.0), LS_K892_rho770_S, SF_K892_rho770_S, 2);
  Amplitude* amp_K892_rho770_P       = new Amplitude( "K892_rho770_P",   new Variable("K892_rho770_P_real",   1.0),   new Variable("K892_rho770_P_imag", 0.0), LS_K892_rho770_P, SF_K892_rho770_P , 2);
  Amplitude* amp_K892_rho770_D       = new Amplitude( "K892_rho770_D",   new Variable("K892_rho770_D_real",    1.0), new Variable("K892_rho770_D_imag",0.0), LS_K892_rho770_D, SF_K892_rho770_D, 2);
  Amplitude* amp_K1410_rho770_P      = new Amplitude( "K1410_rho770",    new Variable("K1410_rho770_P_real",   4.001),  new Variable("K1410_rho770_P_imag",-2.620), LS_K1410_rho770_P, SF_K1410_rho770_P, 2);
  Amplitude* amp_K892_f0_600         = new Amplitude( "K892_f0600",      new Variable("K892_f0600_real",      -0.770),  new Variable("K892_f0600_imag",  -1.530), LS_K892_f0_600, SF_K892_f0_600, 2);
  Amplitude* amp_rho1450_K0_1430     = new Amplitude( "rho1450_K0_1430", new Variable("rho1450_K0_1430_real", -0.110),   new Variable("rho1450_K0_1430_imag",  1.814), LS_rho1450_K0_1430  , SF_rho1450_K0_1430 , 2);
  Amplitude* amp_K1460_K892          = new Amplitude( "K1460_K892",      new Variable("K1460_K892_real",      -0.696),  new Variable("K1460_K892_imag",  0.326), LS_K1460_K892  , SF_K1460_K892 , 2);
  Amplitude* amp_K1460_f0_1370       = new Amplitude( "K1460_f0_1370",   new Variable("K1460_f0_1370_real",   -0.849),  new Variable("K1460_f0_1370_imag",  0.972), LS_K1460_f0_1370  , SF_K1460_f0_1370 , 2);
  Amplitude* amp_K1_1270_K892        = new Amplitude( "K1_1270_K892",    new Variable("K1_1270_K892_real",    0.601),   new Variable("K1_1270_K892_imag",  -0.182), LS_K1_1270_K892  , SF_K1_1270_K892 , 2);
  Amplitude* amp_K1_1270_rho770      = new Amplitude( "K1_1270_rho770",  new Variable("K1_1270_rho770_real",  -1.523),  new Variable("K1_1270_rho770_imag",  1.244), LS_K1_1270_rho770  , SF_K1_1270_rho770 , 2);
  Amplitude* amp_K1_1270_K0_1430     = new Amplitude( "K1_1270_K0_1430", new Variable("K1_1270_K0_1430_real", 0.248),  new Variable("K1_1270_K0_1430_imag",  -0.088), LS_K1_1270_K0_1430  , SF_K1_1270_K0_1430 , 2);
  Amplitude* amp_K1_1400_K892        = new Amplitude( "K1_1400_K892",    new Variable("K1_1400_K892_real",    -0.808),  new Variable("K1_1400_K892_imag",  -0.358), LS_K1_1400_K892  , SF_K1_1400_K892 , 2);
  Amplitude* amp_NonResA_K892        = new Amplitude( "NonResA_K892",    new Variable("NonResA_K892_real",    -15.322), new Variable("NonResA_K892_imag",  -12.089), LS_NonResA_K892, SF_NonResA_K892, 2);
  Amplitude* amp_K2_1430_K892        = new Amplitude( "K2_1430_K892",    new Variable("K2_1430_K892_real",    17.008),  new Variable("K2_1430_K892_imag",  -5.014), LS_K2_1430_K892  , SF_K2_1430_K892 , 2);
  Amplitude* amp_K2_1430_rho770      = new Amplitude( "K2_1430_rho770",  new Variable("K2_1430_rho770_real",  13.039),  new Variable("K2_1430_rho770_imag",  -1.935), LS_K2_1430_rho770  , SF_K2_1430_rho770 , 2);
  Amplitude* amp_a1_rho770           = new Amplitude( "a1_rho770",       new Variable("a1_rho770_real",        -0.639), new Variable("a1_rho770_imag", -6.801), LS_a1_rho770, SF_a1_rho770, 2);
  Amplitude* amp_a1_f0_600           = new Amplitude( "a1_f0_600",       new Variable("a1_f0_600_real",       -0.062),  new Variable("a1_f0_600_imag",  2.872), LS_a1_f0_600  , SF_a1_f0_600 , 2);
  Amplitude* amp_a1_rho770_D         = new Amplitude( "a1_rho770_D",     new Variable("a1_rho770_D_real",     -9.465), new Variable("a1_rho770_D_imag",  15.390), LS_a1_rho770_D, SF_a1_rho770_D, 2);
  Amplitude* amp_nonRes              = new Amplitude( "nonRes",          new Variable("nonRes_real",     -0.265),       new Variable("nonRes_imag",  -0.003), LS_nonRes, SF_nonRes, 2);


  Amplitude* amp_WS_K892_rho770_S       = new Amplitude("WS_K892_rho770_S",   new Variable("WS_K892_rho770_S_real",      1.0),     new Variable("WS_K892_rho770_S_imag",   0.0), LS_K892_rho770_S, SF_K892_rho770_S, 2);
  Amplitude* amp_WS_K892_rho770_P       = new Amplitude("WS_K892_rho770_P",   new Variable("WS_K892_rho770_P_real",      -0.109),   new Variable("WS_K892_rho770_P_imag",   1.653), LS_K892_rho770_P, SF_K892_rho770_P , 2);
  Amplitude* amp_WS_K892_rho770_D       = new Amplitude("WS_K892_rho770_D",   new Variable("WS_K892_rho770_D_real",       25.463), new Variable("WS_K892_rho770_D_imag",     2.662), LS_K892_rho770_D, SF_K892_rho770_D, 2);
  Amplitude* amp_WS_rho1450_K0_1430     = new Amplitude("WS_rho1450_K0_1430", new Variable("WS_rho1450_K0_1430_real",  2.353),   new Variable("WS_rho1450_K0_1430_imag",     -0.234), LS_rho1450_K0_1430  , SF_rho1450_K0_1430 , 2);
  Amplitude* amp_WS_K1_1270_K892        = new Amplitude("WS_K1_1270_K892",    new Variable("WS_K1_1270_K892_real",        -0.035),   new Variable("WS_K1_1270_K892_imag",    -1.405), LS_K1_1270_K892  , SF_K1_1270_K892 , 2);
  Amplitude* amp_WS_K1_1270_rho770      = new Amplitude("WS_K1_1270_rho770",  new Variable("WS_K1_1270_rho770_real",    2.42),  new Variable("WS_K1_1270_rho770_imag",       2.471), LS_K1_1270_rho770  , SF_K1_1270_rho770 , 2);
  Amplitude* amp_WS_K1_1270_K0_1430     = new Amplitude("WS_K1_1270_K0_1430", new Variable("WS_K1_1270_K0_1430_real",  -1.990),  new Variable("WS_K1_1270_K0_1430_imag",     -2.105), LS_K1_1270_K0_1430  , SF_K1_1270_K0_1430 , 2);
  Amplitude* amp_WS_K1_1400_K892        = new Amplitude("WS_K1_1400_K892",    new Variable("WS_K1_1400_K892_real",        -3.347),  new Variable("WS_K1_1400_K892_imag",     -2.237), LS_K1_1400_K892  , SF_K1_1400_K892 , 2);
  Amplitude* amp_WS_nonRes              = new Amplitude("WS_nonRes",            new Variable("WS_nonRes_real",  -0.456), new Variable("WS_nonRes_imag",              -0.041), LS_nonRes, SF_nonRes, 2);

  //DK3P_DI->amplitudes_B.push_back(amp_K892_rho770_S);
  DK3P_DI->amplitudes_B.push_back(amp_K892_rho770_P);
  DK3P_DI->amplitudes_B.push_back(amp_K892_rho770_D);
  //DK3P_DI->amplitudes_B.push_back(amp_K1410_rho770_P);
  //DK3P_DI->amplitudes_B.push_back(amp_K892_f0_600);
  //DK3P_DI->amplitudes_B.push_back(amp_rho1450_K0_1430);
  //DK3P_DI->amplitudes_B.push_back(amp_K1460_K892);
  //DK3P_DI->amplitudes_B.push_back(amp_K1460_f0_1370);
  //DK3P_DI->amplitudes_B.push_back(amp_K1_1270_K892);
  //DK3P_DI->amplitudes_B.push_back(amp_K1_1270_rho770);
  //DK3P_DI->amplitudes_B.push_back(amp_K1_1270_K0_1430);
  //DK3P_DI->amplitudes_B.push_back(amp_K1_1400_K892);
  //DK3P_DI->amplitudes_B.push_back(amp_NonResA_K892);
  //DK3P_DI->amplitudes_B.push_back(amp_K2_1430_K892);
  //DK3P_DI->amplitudes_B.push_back(amp_K2_1430_rho770);
  //DK3P_DI->amplitudes_B.push_back(amp_a1_rho770);
  //DK3P_DI->amplitudes_B.push_back(amp_a1_f0_600);
  //DK3P_DI->amplitudes_B.push_back(amp_a1_rho770_D);
  //DK3P_DI->amplitudes_B.push_back(amp_nonRes);

  //DK3P_DI->amplitudes.push_back(amp_WS_K892_rho770_S);
  //DK3P_DI->amplitudes.push_back(amp_WS_K892_rho770_P);
  //DK3P_DI->amplitudes.push_back(amp_WS_K892_rho770_D);
  //DK3P_DI->amplitudes.push_back(amp_WS_rho1450_K0_1430);
  //DK3P_DI->amplitudes.push_back(amp_WS_K1_1270_K892);
  //DK3P_DI->amplitudes.push_back(amp_WS_K1_1270_rho770);
  //DK3P_DI->amplitudes.push_back(amp_WS_K1_1270_K0_1430);
  //DK3P_DI->amplitudes.push_back(amp_WS_K1_1400_K892);
  //DK3P_DI->amplitudes.push_back(amp_WS_nonRes);

  DK3P_DI->_tau = new Variable("tau", 0.4101);
  DK3P_DI->_xmixing = new Variable("xmixing", 0.0049);
  DK3P_DI->_ymixing = new Variable("ymixing", 0.0061);
  // DK3P_DI->_xmixing = new Variable("xmixing", 0, 0.00001, -.15, .15);
  // DK3P_DI->_ymixing = new Variable("ymixing", 0, 0.00001, -.15, .15);
  DK3P_DI->_SqWStoRSrate = new Variable("SqWStoRSrate", 1.0/sqrt(300.0));  


  Variable* m12 = new Variable("m12", 0, 3);
  Variable* m34 = new Variable("m34", 0, 3); 
  Variable* cos12 = new Variable("cos12", -1, 1);
  Variable* cos34 = new Variable("m12", -1, 1);
  Variable* phi = new Variable("phi", -3.5, 3.5);
  Variable* eventNumber = new Variable("eventNumber", 0, INT_MAX);
  Variable* dtime = new Variable("dtime", 0, 10);
  Variable* sigmat = new Variable("sigmat",-3,3);
  Variable* constantOne = new Variable("constantOne", 1); 
  Variable* constantZero = new Variable("constantZero", 0);
 

  std::vector<Variable*> vars;
  vars.push_back(m12);
  vars.push_back(m34);
  vars.push_back(cos12);
  vars.push_back(cos34);
  vars.push_back(phi);
  vars.push_back(eventNumber); 
  vars.push_back(dtime); 
  vars.push_back(sigmat); 
  UnbinnedDataSet currData(vars); 

 
  DK3P_DI->_xmixing->value = strtof(argv[5], NULL);
  DK3P_DI->_ymixing->value = strtof(argv[6], NULL);

  vector<Variable*> observables;
  vector<Variable*> coefficients; 
  vector<Variable*> offsets;

  observables.push_back(m12);
  observables.push_back(m34);
  observables.push_back(cos12);
  observables.push_back(cos34);
  observables.push_back(phi);
  observables.push_back(eventNumber);
  observables.push_back(dtime);
  observables.push_back(sigmat);
  offsets.push_back(constantZero);
  offsets.push_back(constantZero);
  coefficients.push_back(constantOne); 

  TruthResolution* dat = new TruthResolution();
  PolynomialPdf* eff = new PolynomialPdf("constantEff", observables, coefficients, offsets, 0);
  TDDP4* dp = new TDDP4("test_TD", observables, DK3P_DI, dat, eff, 0, 1);
 
  //dp->setGenDecayTimeLimit(0,3.5); // this corresponds to rougly 97% of the exponential. So this should be ok. And speeds up Generation significantly compared to [0,5] 

  TFile *file = new TFile( argv[4] , "RECREATE");
  TTree *tree = new TTree("events", "events");

  double tm12_2,tm34_2,tc12_2,tc34_2,tphi_2,tdtime_2;

  tree->Branch("m12",          &tm12_2,         "m12/D");
  tree->Branch("m34",          &tm34_2,         "m34/D");
  tree->Branch("c12",          &tc12_2,         "c12/D");
  tree->Branch("c34",          &tc34_2,         "c34/D");
  tree->Branch("phi",          &tphi_2,         "phi/D");
  tree->Branch("dtime",        &tdtime_2,       "dtime/D");

  mcbooster::FlagAcceptReject(0,0);
  int generatedEvents = 0;
  int RunNum = 0;
  int BatchSize = strtoul(argv[1], NULL,0);
  unsigned int offi = strtoul(argv[3], NULL,0);
  unsigned int genEvts =strtoul(argv[2], NULL,0);

  double wmax = 0;
  mcbooster::FlagAcceptReject FlagIt = mcbooster::FlagAcceptReject(0.1,5);

  
  while(generatedEvents < genEvts )
  {
    unsigned int keptEvts = 0;
    dp->setGenerationOffset(offi);
    auto tuple = dp->GenerateSig(BatchSize);
    auto particles = std::get<0>(tuple);
    auto variables = std::get<1>(tuple);
    auto weights = std::get<2>(tuple);
    auto flags = std::get<3>(tuple);
    // int accepted = thrust::count_if(flags.begin(), flags.end(), thrust::identity<bool>());
    ++RunNum;
    // generatedEvents += accepted;
    for (int i = 0; i < weights.size(); ++i)
    {
      if (wmax<weights[i]) wmax = weights[i];
      if (generatedEvents < genEvts && FlagIt(i,weights[i])){
        ++generatedEvents;
        ++keptEvts;
        // printf("PF %i: %s %.5g %.5g %.5g %.5g %.5g %.5g\n",i, (bool)flags[i] ? "true" : "false", weights[i], (*(variables[0]))[i], (*(variables[1]))[i], (*(variables[2]))[i], (*(variables[3]))[i], (*(variables[4]))[i]);
        tm12_2 = (*(variables[0]))[i];
        tm34_2 = (*(variables[1]))[i];
        tc12_2 = (*(variables[2]))[i];
        tc34_2 = (*(variables[3]))[i];
        tphi_2 = (*(variables[4]))[i];
        tdtime_2 = (*(variables[5]))[i];
        tree->Fill();
        // printf("Buffer %i: %.5g %.5g %.5g %.5g %.5g %.5g \n",i, (*myweights)[i],(*Buffer_m12)[i], (*Buffer_m34)[i], (*Buffer_c12)[i], (*Buffer_c34)[i], (*Buffer_phi)[i], (*Buffer_dt)[i]);
      }
    }
    fprintf(stderr,"Run # %i: x=%.6g y=%.6g Using accept-reject method leaves you with %i out of %i events.  %.4g %% of Total offset: %u\n",RunNum, DK3P_DI->_xmixing->value, DK3P_DI->_ymixing->value, keptEvts, BatchSize, generatedEvents*100.0/genEvts, offi);
    offi += BatchSize;
    delete variables[0];
    delete variables[1];
    delete variables[2];
    delete variables[3];
    delete variables[4];
    delete variables[5];

    delete particles[0];
    delete particles[1];
    delete particles[2];
    delete particles[3];
  }
  // printf("start\n");
  // int i = 0;
  // printf("Buffer %i: %.5g %.5g %.5g %.5g %.5g %.5g \n",i, (*myweights)[i],(*Buffer_m12)[i], (*Buffer_m34)[i], (*Buffer_c12)[i], (*Buffer_c34)[i], (*Buffer_phi)[i], (*Buffer_dt)[i]);

  // printf("start2\n");
  std::ofstream out;
  string outname ="Max_observed_weights.txt";
  out.open(outname.c_str(), std::ios::app);
  out.precision(10);

  out << wmax <<endl;

  tree->Write();
  file->Close();
  // printf("overall wmax %f, keept %u evts, reweight ratio %.5g\n",wmax, keptEvts, (double)keptEvts/genEvts );
  printf("%i\n",offi);
  return 0; 

}
